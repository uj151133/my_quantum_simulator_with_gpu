#include "hip/hip_runtime.h"
#include "state.cuh"

__global__ void createKet0Node(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
}

__global__ void createKet1Node(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(1.0, nullptr);
}

__global__ void createKetPlusNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(1.0, nullptr);
}

__global__ void createKetMinusNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(-1.0, nullptr);
}

__global__ void createBra0Node(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
}

__global__ void createBra0Node(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(1.0, nullptr);
}

__global__ void createBraPlusNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(1.0, nullptr);
}

__global__ void createBraMinusNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(-1.0, nullptr);
}
/////////////////////////////////////
//
//	KET VECTORS
//
/////////////////////////////////////

QMDDState state::KetO() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createKet0Node<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge ket0Edge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(ket0Edge);
}

QMDDState state::Ket1() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createKet1Node<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge ket1Edge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(ket1Edge);
}

QMDDState state::KetPlus() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createKetPlusNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge ketPlusEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(ketPlusEdge);
}

QMDDState state::KetMinus() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createKetMinusNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge ketMinusEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(ketMinusEdge);
}

QMDDState state::Bra0() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createBra0Node<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge bra0Edge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(bra0Edge);
}

QMDDState state::Bra1() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createBra1Node<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge bra1Edge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(bra1Edge);
}

QMDDState state::BraPlus() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createBraPlusNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge braPlusEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(braPlusEdge);
}

QMDDState state::BraMinus() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createBraPlusNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge braMinusEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDState(braMinusEdge);
}