#include "hip/hip_runtime.h"
#include "gate.cuh"


__device__ hipDoubleComplex i = make_hipDoubleComplex(0.0, 1.0);

__global__ void createZeroNode(QMDDNode* node) {
    node->edges.push_back(QMDDEdge(0.0, nullptr));
    node->edges.push_back(QMDDEdge(0.0, nullptr));
    node->edges.push_back(QMDDEdge(0.0, nullptr));
    node->edges.push_back(QMDDEdge(0.0, nullptr));
}

__global__ void createIdentityGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createGlobalPhaseGate(hipDoubleComplex* weights, hipDoubleComplex** nodes, double delta) {
    weights[0] = cuCexp(hipCmul(i, make_hipDoubleComplex(delta, 0.0)));
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createPauliXGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0
    nodes[0]->edges[0] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(0.0, nullptr);
}

__global__ void createPauliYGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = i;
    nodes[0]->edges[0] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(0.0, nullptr);
}

__global__ void createPauliZGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseSGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(i, nullptr);
}

__global__ void createSquareRootOfXGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0 / 2.0 + i / 2.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(i, nullptr);
    nodes[0]->edges[2] = QMDDEdge(i, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createHadamardGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseShiftGate(hipDoubleComplex* weights, hipDoubleComplex** nodes, double phi) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(phi, 0.0))), nullptr);
}

__global__ void createPhaseTGate(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(M_PI / 4.0, 0.0))), nullptr);
}

__global__ void createRotationAboutXGate(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[1] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[2] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[3] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void create RotationAboutYGate(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[1] = QMDDEdge(-sin(theta / 2.0), nullptr);
    nodes[0]->edges[2] = QMDDEdge(sin(theta / 2.0), nullptr);
    nodes[0]->edges[3] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void createRotationAboutZGate(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cuCexp(hipCmul(-i, make_hipDoubleComplex(theta / 2.0, 0.0))), nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(theta / 2.0, 0.0))), nullptr);
}

QMDDGate gate::ZERO() {
    QMDDNode* zeroNode;
    hipMalloc(&zeroNode, sizeof(QMDDNode));
    createZeroNode<<<1, 1>>>(zeroNode);

    QMDDGate zeroGate(QMDDEdge(0.0, zeroNode));
    hipFree(zeroNode);

    return zeroGate;
}

QMDDGate gate::I() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createIdentityGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge iEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(iEdge);
}


QMDDGate gate::Ph(double delta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createGlobalPhaseGate<<<1, 1>>>(weights, nodes, delta);
    hipDeviceSynchronize();
    
    QMDDEdge phEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(phEdge);
}

QMDDGate gate::X() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliXGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge xEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(xEdge);
}

QMDDGate gate::Y() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliYGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge yEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(yEdge);
}

QMDDGate gate::Z() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliZGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge zEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(zEdge);
}

QMDDGate gate::S() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseSGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge sEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(sEdge);
}

QMDDGate gate::V() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createSquareRootOfXGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge vEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(vEdge);
}

QMDDGate gate::H() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createHadamardGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge hEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(hEdge);
}

QMDDGate gate::P(double phi) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseShiftGate<<<1, 1>>>(weights, nodes, phi);
    hipDeviceSynchronize();
    
    QMDDEdge pEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(pEdge);
}

QMDDGate gate::T() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseTGate<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge tEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(tEdge);
}

QMDDGate gate::Rx(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutXGate<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Ry(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutYGate<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Rz(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutZGate<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rzEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rzEdge);
}

QMDDGate Ph(double delta) {
    hipDoubleComplex* weights;
    QMDDNode** nodes;
    
    // メモリの確保
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));

    // カーネルの呼び出し
    createPhaseGate<<<1, 1>>>(weights, nodes, delta);
    hipDeviceSynchronize();

    // QMDDEdge の作成
    QMDDEdge phEdge(weights[0], nodes[0]);
    
    // メモリの解放
    hipFree(nodes[0]);  // ノードのメモリ解放
    hipFree(nodes);     // ノード配列のメモリ解放
    hipFree(weights);   // ウェイトのメモリ解放

    // QMDDGate の作成
    return QMDDGate(phEdge);
}
