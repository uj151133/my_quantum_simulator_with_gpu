#include "hip/hip_runtime.h"
#include "gate.cuh"


__device__ hipDoubleComplex i = make_hipDoubleComplex(.0, 1.0);

__global__ void createZeroNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = .0;
    nodes[0]->edges[0][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(.0, nullptr);
}

__global__ void createIdentityNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(1.0, nullptr);
}

__global__ void createGlobalPhaseNode(hipDoubleComplex* weights, QMDDNode* nodes, double delta) {
    weights[0] = cuCexp(hipCmul(i, make_hipDoubleComplex(delta, .0)));
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(1.0, nullptr);
}

__global__ void createPauliXNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0
    nodes[0]->edges[0][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(.0, nullptr);
}

__global__ void createPauliYNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = i;
    nodes[0]->edges[0][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(.0, nullptr);
}

__global__ void createPauliZNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseSNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(i, nullptr);
}

__global__ void createSquareRootOfXNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0 / 2.0 + i / 2.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(i, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(i, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(1.0, nullptr);
}

__global__ void createHadamardNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseShiftNode(hipDoubleComplex* weights, QMDDNode* nodes, double phi) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(phi, .0))), nullptr);
}

__global__ void createPhaseTNode(hipDoubleComplex* weights, QMDDNode* nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(M_PI / 4.0, .0))), nullptr);
}

__global__ void createRotationAboutXNode(hipDoubleComplex* weights, QMDDNode* nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void create RotationAboutYNode(hipDoubleComplex* weights, QMDDNode* nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(-sin(theta / 2.0), nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(sin(theta / 2.0), nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void createRotationAboutZNode(hipDoubleComplex* weights, QMDDNode* nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0][0] = QMDDEdge(cuCexp(hipCmul(-i, make_hipDoubleComplex(theta / 2.0, .0))), nullptr);
    nodes[0]->edges[0][1] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][0] = QMDDEdge(.0, nullptr);
    nodes[0]->edges[1][1] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(theta / 2.0, .0))), nullptr);
}

QMDDGate gate::O() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createZeroNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge zeroEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(zeroEdge);
}

QMDDGate gate::I() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createIdentityNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge iEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(iEdge);
}


QMDDGate gate::Ph(double delta) {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createGlobalPhaseNode<<<1, 1>>>(weights, nodes, delta);
    hipDeviceSynchronize();
    
    QMDDEdge phEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(phEdge);
}

QMDDGate gate::X() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliXNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge xEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(xEdge);
}

QMDDGate gate::Y() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliYNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge yEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(yEdge);
}

QMDDGate gate::Z() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliZNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge zEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(zEdge);
}

QMDDGate gate::S() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseSNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge sEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(sEdge);
}

QMDDGate gate::V() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createSquareRootOfXNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge vEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(vEdge);
}

QMDDGate gate::H() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createHadamardNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge hEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(hEdge);
}

QMDDGate gate::P(double phi) {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseShiftNode<<<1, 1>>>(weights, nodes, phi);
    hipDeviceSynchronize();
    
    QMDDEdge pEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(pEdge);
}

QMDDGate gate::T() {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseTNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge tEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(tEdge);
}

QMDDGate gate::Rx(double theta) {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutXNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Ry(double theta) {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutYNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Rz(double theta) {
    hipDoubleComplex* weights;
    QMDDNode* nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutZNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rzEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rzEdge);
}

