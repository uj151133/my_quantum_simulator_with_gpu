#include "hip/hip_runtime.h"
#include "gate.cuh"


__device__ hipDoubleComplex i = make_hipDoubleComplex(0.0, 1.0);

__global__ void createZeroNode(QMDDNode* node) {
    weights[0] = 0.0;
    nodes[0]->edges[0] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(0.0, nullptr);
}

__global__ void createIdentityNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weights[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createGlobalPhaseNode(hipDoubleComplex* weights, hipDoubleComplex** nodes, double delta) {
    weights[0] = cuCexp(hipCmul(i, make_hipDoubleComplex(delta, 0.0)));
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createPauliXNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0
    nodes[0]->edges[0] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(0.0, nullptr);
}

__global__ void createPauliYNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = i;
    nodes[0]->edges[0] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(0.0, nullptr);
}

__global__ void createPauliZNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseSNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(i, nullptr);
}

__global__ void createSquareRootOfXNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0 / 2.0 + i / 2.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(i, nullptr);
    nodes[0]->edges[2] = QMDDEdge(i, nullptr);
    nodes[0]->edges[3] = QMDDEdge(1.0, nullptr);
}

__global__ void createHadamardNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0 / sqrt(2.0);
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(-1.0, nullptr);
}

__global__ void createPhaseShiftNode(hipDoubleComplex* weights, hipDoubleComplex** nodes, double phi) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(phi, 0.0))), nullptr);
}

__global__ void createPhaseTNode(hipDoubleComplex* weights, hipDoubleComplex** nodes) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(1.0, nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(M_PI / 4.0, 0.0))), nullptr);
}

__global__ void createRotationAboutXNode(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[1] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[2] = QMDDEdge(-i * sin(theta / 2.0), nullptr);
    nodes[0]->edges[3] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void create RotationAboutYNode(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cos(theta / 2.0), nullptr);
    nodes[0]->edges[1] = QMDDEdge(-sin(theta / 2.0), nullptr);
    nodes[0]->edges[2] = QMDDEdge(sin(theta / 2.0), nullptr);
    nodes[0]->edges[3] = QMDDEdge(cos(theta / 2.0), nullptr);
}

__global__ void createRotationAboutZNode(hipDoubleComplex* weights, hipDoubleComplex** nodes, double theta) {
    weighs[0] = 1.0;
    nodes[0]->edges[0] = QMDDEdge(cuCexp(hipCmul(-i, make_hipDoubleComplex(theta / 2.0, 0.0))), nullptr);
    nodes[0]->edges[1] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[2] = QMDDEdge(0.0, nullptr);
    nodes[0]->edges[3] = QMDDEdge(cuCexp(hipCmul(i, make_hipDoubleComplex(theta / 2.0, 0.0))), nullptr);
}

QMDDGate gate::ZERO() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createZeroNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge zeroEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(zeroEdge);
}

QMDDGate gate::I() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createIdentityNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge iEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(iEdge);
}


QMDDGate gate::Ph(double delta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createGlobalPhaseNode<<<1, 1>>>(weights, nodes, delta);
    hipDeviceSynchronize();
    
    QMDDEdge phEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(phEdge);
}

QMDDGate gate::X() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliXNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge xEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(xEdge);
}

QMDDGate gate::Y() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliYNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge yEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(yEdge);
}

QMDDGate gate::Z() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPauliZNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge zEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(zEdge);
}

QMDDGate gate::S() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseSNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge sEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(sEdge);
}

QMDDGate gate::V() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createSquareRootOfXNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge vEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(vEdge);
}

QMDDGate gate::H() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createHadamardNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge hEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(hEdge);
}

QMDDGate gate::P(double phi) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseShiftNode<<<1, 1>>>(weights, nodes, phi);
    hipDeviceSynchronize();
    
    QMDDEdge pEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(pEdge);
}

QMDDGate gate::T() {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createPhaseTNode<<<1, 1>>>(weights, nodes);
    hipDeviceSynchronize();
    
    QMDDEdge tEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(tEdge);
}

QMDDGate gate::Rx(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutXNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Ry(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutYNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rxEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rxEdge);
}

QMDDGate gate::Rz(double theta) {
    hipDoubleComplex* weights;
    hipDoubleComplex** nodes;
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));
    
    createRotationAboutZNode<<<1, 1>>>(weights, nodes, theta);
    hipDeviceSynchronize();
    
    QMDDEdge rzEdge(weights[0], nodes[0]);

    hipFree(weights);
    hipFree(nodes[0]);
    hipFree(nodes);

    return QMDDGate(rzEdge);
}

QMDDGate Ph(double delta) {
    hipDoubleComplex* weights;
    QMDDNode** nodes;
    
    // メモリの確保
    hipMallocManaged(&weights, sizeof(hipDoubleComplex) * 1);
    hipMallocManaged(&nodes, sizeof(QMDDNode*) * 1);
    hipMallocManaged(&nodes[0], sizeof(QMDDNode));

    // カーネルの呼び出し
    createPhaseNode<<<1, 1>>>(weights, nodes, delta);
    hipDeviceSynchronize();

    // QMDDEdge の作成
    QMDDEdge phEdge(weights[0], nodes[0]);
    
    // メモリの解放
    hipFree(nodes[0]);  // ノードのメモリ解放
    hipFree(nodes);     // ノード配列のメモリ解放
    hipFree(weights);   // ウェイトのメモリ解放

    // QMDDGate の作成
    return QMDDGate(phEdge);
}
